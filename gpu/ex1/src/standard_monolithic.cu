
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void vectorAdd(int n, float *a, float *b, float *c)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < n) 
      c[i] = a[i] + b[i];
}

int main(void)
{
  int N = pow(2, 24);
  float *a, *b, *c, *d_a, *d_b, *d_c;
  a = (float*)malloc(N*sizeof(float));
  b = (float*)malloc(N*sizeof(float));
  c = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_a, N*sizeof(float));
  hipMalloc(&d_b, N*sizeof(float));
  hipMalloc(&d_c, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
    b[i] = 2.0f;
    c[i] = 0.0f;
  }

  hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, N*sizeof(float), hipMemcpyHostToDevice);

  vectorAdd<<<1, 32>>>(N, a, b, c);

  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(c[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(a);
  hipFree(b);
  hipFree(c);
  
  return 0;
}